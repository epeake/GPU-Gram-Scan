#include "hip/hip_runtime.h"
#include "cuda-util.h"
#include "gpu_graham_scan.h"

/*
 * see if p1 is "less" than p2
 */
template <class Num_Type>
__device__ bool comparePoints(const gpu_graham_scan::Point<Num_Type> p1,
                              const gpu_graham_scan::Point<Num_Type> p2) {
  // cross product of 2 points
  Num_Type x_product = (p1.x_ * p2.y_) - (p2.x_ * p1.y_);

  if (x_product > 0) {
    return true;
  }
  if (x_product == 0) {
    Num_Type sq_mag_p1 = (p1.x_ * p1.x_) + (p1.y_ * p1.y_);
    Num_Type sq_mag_p2 = (p2.x_ * p2.x_) + (p2.y_ * p2.y_);
    return sq_mag_p1 < sq_mag_p2;
  }
  return false;
}

template <class Num_Type>
__global__ void BuildBitonicKernel(size_t n_points,
                                   gpu_graham_scan::Point<Num_Type>* d_points,
                                   size_t threads_per_chunk, size_t chunk_len) {
  size_t true_idx = threadIdx.x + (blockIdx.x * blockDim.x);
  size_t chunk_offset = (true_idx / threads_per_chunk) * chunk_len;
  size_t thread_offset = true_idx % threads_per_chunk;
  size_t first = thread_offset + chunk_offset;
  size_t last = (chunk_offset + chunk_len - 1) - thread_offset;
  if (last < n_points && comparePoints(d_points[last], d_points[first])) {
    gpu_graham_scan::Point<Num_Type> tmp = d_points[last];
    d_points[last] = d_points[first];
    d_points[first] = tmp;
  } else if (last < n_points &&
             !comparePoints(d_points[last], d_points[first])) {
  }
}

template <class Num_Type>
__global__ void BitonicSortKernel(size_t n_points,
                                  gpu_graham_scan::Point<Num_Type>* d_points,
                                  size_t threads_per_chunk, size_t chunk_len) {
  size_t true_idx = threadIdx.x + (blockIdx.x * blockDim.x);
  size_t chunk_offset = (true_idx / threads_per_chunk) * chunk_len;
  size_t thread_offset = true_idx % threads_per_chunk;
  size_t first = thread_offset + chunk_offset;
  size_t last = first + threads_per_chunk;
  if (last < n_points && comparePoints(d_points[last], d_points[first])) {
    gpu_graham_scan::Point<Num_Type> tmp = d_points[last];
    d_points[last] = d_points[first];
    d_points[first] = tmp;
  } else if (last < n_points &&
             !comparePoints(d_points[last], d_points[first])) {
  }
}

template <class Num_Type>
void gpu_graham_scan::BitonicSortPoints(
    gpu_graham_scan::Point<Num_Type>* points_arr, size_t n_points) {
  const uint threads_per_block = 1024;  // Max threads_per_block = 1024;

  // Allocate device data
  gpu_graham_scan::Point<Num_Type>* d_points;

  cudaErrorCheck(hipMalloc(
      &d_points, n_points * sizeof(gpu_graham_scan::Point<Num_Type>)));

  // points to device
  cudaErrorCheck(hipMemcpy(d_points, points_arr,
                            n_points * sizeof(gpu_graham_scan::Point<Num_Type>),
                            hipMemcpyHostToDevice));

  // round up to the the power of 2 to get our upper bound
  size_t upper_bound = n_points;
  uint power = 0;
  while (upper_bound) {
    upper_bound >>= 1;
    power++;
  }
  size_t curr_bound = 1 << (power - 1);
  upper_bound = (curr_bound < n_points) ? (curr_bound << 1) : curr_bound;

  size_t total_threads = upper_bound >> 1;
  for (size_t i = 2, j = i; i <= upper_bound; i *= 2, j = i) {
    size_t threads_per_chunk = j >> 1;
    BuildBitonicKernel<<<(total_threads + threads_per_block - 1) /
                             threads_per_block,
                         threads_per_block>>>(n_points, d_points,
                                              threads_per_chunk, j);

    // wait for build to finish
    cudaErrorCheck(hipDeviceSynchronize());
    j >>= 1;
    while (j > 1) {
      threads_per_chunk = j >> 1;
      BitonicSortKernel<<<(total_threads + threads_per_block - 1) /
                              threads_per_block,
                          threads_per_block>>>(n_points, d_points,
                                               threads_per_chunk, j);
      cudaErrorCheck(hipDeviceSynchronize());
      j >>= 1;
    }
  }

  // Copy points back to host points to device
  cudaErrorCheck(hipMemcpy(points_arr, d_points,
                            n_points * sizeof(gpu_graham_scan::Point<Num_Type>),
                            hipMemcpyDeviceToHost));

  // Cleanup device data
  cudaErrorCheck(hipFree(d_points));
}

/*
 * All the implementations...
 */
template void gpu_graham_scan::BitonicSortPoints(
    gpu_graham_scan::Point<int32_t>* points_arr, size_t n_points);

template void gpu_graham_scan::BitonicSortPoints(
    gpu_graham_scan::Point<float>* points_arr, size_t n_points);

template void __global__
BuildBitonicKernel(size_t n_points, gpu_graham_scan::Point<int32_t>* d_points,
                   size_t threads_per_chunk, size_t chunk_len);

template void __global__
BuildBitonicKernel(size_t n_points, gpu_graham_scan::Point<float>* d_points,
                   size_t threads_per_chunk, size_t chunk_len);

template __global__ void BitonicSortKernel(
    size_t n_points, gpu_graham_scan::Point<int32_t>* d_points,
    size_t threads_per_chunk, size_t chunk_len);

template __global__ void BitonicSortKernel(
    size_t n_points, gpu_graham_scan::Point<float>* d_points,
    size_t threads_per_chunk, size_t chunk_len);

template __device__ bool comparePoints(
    const gpu_graham_scan::Point<int32_t> p1,
    const gpu_graham_scan::Point<int32_t> p2);

template __device__ bool comparePoints(const gpu_graham_scan::Point<float> p1,
                                       const gpu_graham_scan::Point<float> p2);
