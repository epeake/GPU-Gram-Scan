#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "cuda-util.h"
#include "gpu_graham_scan.h"

const uint kMaxThreads = 1024;
const uint kTotalSMs = 46;
const uint kMaxThreadsSpan = kMaxThreads * kTotalSMs * 2;

template <class Num_Type>
__global__ void BuildBitonic(size_t start_size) {
  while (start_size > 1) {
    start_size >>= 1;
  }
}

template <class Num_Type>
void gpu_graham_scan::BitonicSortPoints(
    std::vector<gpu_graham_scan::Point<Num_Type>> points) {
  size_t n_points = points.size();

  // underlying array of points to put onto GPU
  gpu_graham_scan::Point<Num_Type>* points_arr = points.data();

  // Allocate device data
  Num_Type* d_points;

  cudaErrorCheck(hipMalloc(
      &d_points, n_points * sizeof(gpu_graham_scan::Point<Num_Type>)));

  // points to device
  cudaErrorCheck(hipMemcpy(d_points, points_arr,
                            n_points * sizeof(gpu_graham_scan::Point<Num_Type>),
                            hipMemcpyHostToDevice));

  // round up to the the power of 2 to get our upper bound
  size_t upper_bound = n_points;
  uint power = 0;
  while (upper_bound) {
    upper_bound >>= 1;
    power++;
  }
  size_t curr_bound = 1 << (power - 1);
  upper_bound = (curr_bound < n_points) ? (curr_bound << 1) : curr_bound;

  for (size_t i = 2, j = i; i <= upper_bound; i *= 2, j = i) {
    size_t chunks;
    size_t threads_per_chunk;
    if (j > kMaxThreadsSpan) {
      chunks = 1;
      threads_per_chunk = 1;
    } else {
      chunks = (n_points + j - 1) / j;
      threads_per_chunk = j >> 1;
    }

    // each chunk thread until chunk size == ...

    // BuildBitonic<<<>>>;
    j >>= 1;
    while (j > 1) {
      // sort bionic
      // BitonicSortPointsKernel<<<dim3(BX, BY), dim3(TX, TY)>>>();
      j >>= 1;
    }
  }

  // Copy points back to host points to device
  cudaErrorCheck(hipMemcpy(points_arr, d_points,
                            n_points * sizeof(gpu_graham_scan::Point<Num_Type>),
                            hipMemcpyDeviceToHost));

  // Cleanup device data
  cudaErrorCheck(hipFree(d_points));
}

template void gpu_graham_scan::BitonicSortPoints(
    std::vector<gpu_graham_scan::Point<int>> points);

template void gpu_graham_scan::BitonicSortPoints(
    std::vector<gpu_graham_scan::Point<double>> points);
